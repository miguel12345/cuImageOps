#include "hip/hip_runtime.h"
#include "utils.cu"

extern "C" __global__ 
void scale(float* image,float* out, float* scale, float* pivot, unsigned int* dims, unsigned int fillMode)
{
 size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
 size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

 unsigned int height = dims[0];
 unsigned int width = dims[1];
 unsigned int channels = dims[2];

 float pivotx = pivot[0] * (width-1);
 float pivoty = pivot[1] * (height-1);

 float srcx = ((float(dstx) - pivotx)/ scale[0]) + pivotx;
 float srcy = ((float(dsty) - pivoty)/ scale[1]) + pivoty;


 if (dstx >= width || dsty >= height)
    return;

  size_t outIdx = dsty*width + dstx;

  if(channels == 3){
    float3* image3c = (float3*)&image[0];
    float3* out3c = (float3*)&out[0];
    out3c[outIdx] = sample2d<float3>(image3c,srcx,srcy,dims,fillMode,INTERPOLATION_MODE_POINT,make_float3(0.0f,0.0f,0.0f));
  }
  else{
    out[outIdx] = sample2d<float>(image,srcx,srcy,dims,fillMode,INTERPOLATION_MODE_POINT,0.0f);
  }

  
}