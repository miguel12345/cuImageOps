
#include <hip/hip_runtime.h>
#define FILL_MODE_CONSTANT 1
#define FILL_MODE_REFLECTION 2

#define INTERPOLATION_MODE_POINT 1
#define INTERPOLATION_MODE_LINEAR 2

inline __device__ float3 operator*(float3 a,float b) {
    return make_float3(a.x*b,a.y*b,a.z*b);
}

inline __device__ float3 operator+(float3 a,float3 b) {
    return make_float3(a.x+b.x,a.y+b.y,a.z+b.z);
}

__device__ float radians(float a)
{
  return 0.017453292 * a;
}

__device__ float2 rotate(float2 point, float theta){
    float thetarad = radians(theta);
    float thetasin = sin(thetarad);
    float thetacos = cos(thetarad);

    float rotX = point.x * thetacos - point.y * thetasin;
    float rotY = point.y * thetacos + point.x * thetasin;

    return make_float2(rotX,rotY);
}

template<typename type> __device__ type pointsample2d(type* image, float x, float y, unsigned int* dims, unsigned int fillMode,type fillConstant)
{

    unsigned int height = dims[0];
    unsigned int width = dims[1];

    int xInt = __float2int_rn(x);
    int yInt = __float2int_rn(y);


    if(xInt < 0 || xInt >= width || yInt < 0 || yInt >= height) {
        if (fillMode == FILL_MODE_CONSTANT) {
            return fillConstant;
        }
        else if (fillMode == FILL_MODE_REFLECTION)
        {
            if(xInt < 0) {
                xInt = -xInt;
                xInt = (xInt-1)%width;
            }
            else if (xInt >= width)
            {
                xInt = (xInt-1)%width;
                xInt = (width-1) - xInt;
            }
            
            if(yInt < 0) {
                yInt = -yInt;
                yInt = (yInt-1)%height;
                yInt = (height-1) - yInt;
            }
            else if (yInt >= height)
            {
                yInt = (yInt-1)%height;
            }
        }
    }
    
    return image[yInt*width + xInt];
}


template<typename type> __device__ type bilinearsample2d(type* image, float x, float y, unsigned int* dims, unsigned int fillMode, type fillConstant)
{

    

    //Determine the four corners

    float2 tl = make_float2(floorf(x),floorf(y));
    float2 tr = make_float2(floorf(x+1),floorf(y));
    float2 bl = make_float2(floorf(x),floorf(y+1));
    float2 br = make_float2(floorf(x+1),floorf(y+1));

    //Sample the four corners
    
    type tlval = pointsample2d(image,tl.x,tl.y,dims,fillMode,fillConstant);
    type trval = pointsample2d(image,tr.x,tr.y,dims,fillMode,fillConstant);
    type blval = pointsample2d(image,bl.x,bl.y,dims,fillMode,fillConstant);
    type brval = pointsample2d(image,br.x,br.y,dims,fillMode,fillConstant);

    float area = (tr.x-bl.x)*(bl.y-tl.y);

    //Calculate interpolation weights

    float wtl = ((br.x-x)*(br.y-y))/area;
    float wtr = ((x-bl.x)*(bl.y-y))/area;
    float wbl = ((tr.x-x)*(y-tr.y))/area;
    float wbr = ((x-tl.x)*(y-tl.y))/area;

    //Return interpolated result
    return tlval*wtl + trval*wtr + blval*wbl + brval*wbr;

}

template<typename type> __device__ type sample2d(type* image, float x, float y, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode, type fillConstant) {

    if(interpolationMode == INTERPOLATION_MODE_POINT) {
        return pointsample2d(image,x,y,dims,fillMode,fillConstant);
    }
    else if(interpolationMode == INTERPOLATION_MODE_LINEAR) {
        return bilinearsample2d(image,x,y,dims,fillMode,fillConstant);
    }
}