#include "hip/hip_runtime.h"
#include "utils.cu"

#define KERNEL_2D 1
#define KERNEL_1D_HORIZONTAL 2
#define KERNEL_1D_VERTICAL 3

template<typename input_type,typename output_type, typename accum_type> __device__ void kernelSampleAndAssign(input_type* input, output_type* output,unsigned int* dims,uint2 dstPoint, float* kernel, unsigned int kernelSize,unsigned int kernelType, unsigned int fillMode,unsigned int interpolationMode, input_type fillConstant) {

  int halfKernelSize = (kernelSize-1)/2;
  unsigned int kernelCellIdx = 0;

  unsigned int width = dims[1];

  unsigned int outputIdx = dstPoint.y*width + dstPoint.x;

  accum_type aggregatedVal = accum_type();


  if(kernelType == KERNEL_2D) {
    for(int kernelCellX = -halfKernelSize; kernelCellX <= halfKernelSize; kernelCellX++){
        for(int kernelCellY = -halfKernelSize; kernelCellY <= halfKernelSize; kernelCellY++){
            
            float kernelWeight = kernel[kernelCellIdx];

            input_type kernelCellVal = sample2d(input, (float)((int)dstPoint.x + kernelCellX), (float)((int)dstPoint.y + kernelCellY),dims,fillMode,interpolationMode,fillConstant);

            aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;

            kernelCellIdx += 1;
        }
      }
  }
  else if(kernelType == KERNEL_1D_HORIZONTAL) {
    for(int kernelCellX = -halfKernelSize; kernelCellX <= halfKernelSize; kernelCellX++){
          float kernelWeight = kernel[kernelCellIdx];
          input_type kernelCellVal = sample2d(input, (float)((int)dstPoint.x + kernelCellX), (float)dstPoint.y,dims,fillMode,interpolationMode,fillConstant);
          aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;
          kernelCellIdx += 1;
      }
  }
  else if(kernelType == KERNEL_1D_VERTICAL) {
    for(int kernelCellY = -halfKernelSize; kernelCellY <= halfKernelSize; kernelCellY++){
          float kernelWeight = kernel[kernelCellIdx];
          input_type kernelCellVal = sample2d(input, (float)dstPoint.x, (float)((int)dstPoint.y + kernelCellY),dims,fillMode,interpolationMode,fillConstant);
          aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;
          kernelCellIdx += 1;
      }
  }
  

  output[outputIdx] = convert<accum_type,output_type>(aggregatedVal);

}

extern "C" __global__ 
void blur(unsigned char* output, unsigned char* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign<unsigned char,unsigned char, float>(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,0);
  }
  else if(channels == 3){
    
    uchar3* input3c = (uchar3*)&input[0];
    uchar3* output3c = (uchar3*)&output[0];

    kernelSampleAndAssign<uchar3,uchar3, float3>(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,make_uchar3(0,0,0));
  }
  else if(channels == 4){

    uchar4* input4c = (uchar4*)&input[0];
    uchar4* output4c = (uchar4*)&output[0];

    kernelSampleAndAssign<uchar4,uchar4, float4>(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,make_uchar4(0,0,0,0));
    
  }
  
}

extern "C" __global__ void blurHorizontal(float* output,unsigned char* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign<unsigned char,float,float>(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,0);
  }
  else if(channels == 3){
    
    uchar3* input3c = (uchar3*)&input[0];
    float3* output3c = (float3*)&output[0];

    kernelSampleAndAssign<uchar3,float3,float3>(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,make_uchar3(0,0,0));
  }
  else if(channels == 4){

    uchar4* input4c = (uchar4*)&input[0];
    float4* output4c = (float4*)&output[0];

    kernelSampleAndAssign<uchar4,float4,float4>(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,make_uchar4(0,0,0,0));
    
  }
}

extern "C" __global__ void blurVertical(unsigned char* output,float* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign<float,unsigned char,float>(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,0);
  }
  else if(channels == 3){
    
    float3* input3c = (float3*)&input[0];
    uchar3* output3c = (uchar3*)&output[0];

    kernelSampleAndAssign<float3,uchar3,float3>(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,make_float3(0,0,0));
  }
  else if(channels == 4){

    float4* input4c = (float4*)&input[0];
    uchar4* output4c = (uchar4*)&output[0];

    kernelSampleAndAssign<float4,uchar4,float4>(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,make_float4(0,0,0,0));
    
  }
}