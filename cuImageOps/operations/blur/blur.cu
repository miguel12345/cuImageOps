#include "hip/hip_runtime.h"
#include "utils.cu"

#define KERNEL_2D 1
#define KERNEL_1D_HORIZONTAL 2
#define KERNEL_1D_VERTICAL 3

template<typename type> __device__ void kernelSampleAndAssign(type* input, type* output,unsigned int* dims,uint2 dstPoint, float* kernel, unsigned int kernelSize,unsigned int kernelType, unsigned int fillMode,unsigned int interpolationMode, type fillConstant) {

  int halfKernelSize = (kernelSize-1)/2;
  unsigned int kernelCellIdx = 0;

  unsigned int height = dims[0];  
  unsigned int width = dims[1];

  unsigned int outputIdx = dstPoint.y*width + dstPoint.x;

  type aggregatedVal = type();


  if(kernelType == KERNEL_2D) {
    for(int kernelCellX = -halfKernelSize; kernelCellX <= halfKernelSize; kernelCellX++){
        for(int kernelCellY = -halfKernelSize; kernelCellY <= halfKernelSize; kernelCellY++){
            
            float kernelWeight = kernel[kernelCellIdx];

            type kernelCellVal = sample2d(input, (float)((int)dstPoint.x + kernelCellX), (float)((int)dstPoint.y + kernelCellY),dims,fillMode,interpolationMode,fillConstant);

            aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;

            kernelCellIdx += 1;
        }
      }
  }
  else if(kernelType == KERNEL_1D_HORIZONTAL) {
    for(int kernelCellX = -halfKernelSize; kernelCellX <= halfKernelSize; kernelCellX++){
          float kernelWeight = kernel[kernelCellIdx];
          type kernelCellVal = sample2d(input, (float)((int)dstPoint.x + kernelCellX), (float)dstPoint.y,dims,fillMode,interpolationMode,fillConstant);
          aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;
          kernelCellIdx += 1;
      }
  }
  else if(kernelType == KERNEL_1D_VERTICAL) {
    for(int kernelCellY = -halfKernelSize; kernelCellY <= halfKernelSize; kernelCellY++){
          float kernelWeight = kernel[kernelCellIdx];
          type kernelCellVal = sample2d(input, (float)dstPoint.x, (float)((int)dstPoint.y + kernelCellY),dims,fillMode,interpolationMode,fillConstant);
          aggregatedVal = aggregatedVal + kernelCellVal * kernelWeight;
          kernelCellIdx += 1;
      }
  }
  

  output[outputIdx] = aggregatedVal;

}

extern "C" __global__ 
void blur(float* output, float* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,0.0f);
  }
  else if(channels == 3){
    
    float3* input3c = (float3*)&input[0];
    float3* output3c = (float3*)&output[0];

    kernelSampleAndAssign(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,make_float3(0.0,0.0,0.0));
  }
  else if(channels == 4){

    float4* input4c = (float4*)&input[0];
    float4* output4c = (float4*)&output[0];

    kernelSampleAndAssign(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_2D,fillMode,interpolationMode,make_float4(0.0,0.0,0.0,0.0));
    
  }
  
}

extern "C" __global__ void blurHorizontal(float* output,float* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,0.0f);
  }
  else if(channels == 3){
    
    float3* input3c = (float3*)&input[0];
    float3* output3c = (float3*)&output[0];

    kernelSampleAndAssign(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,make_float3(0.0,0.0,0.0));
  }
  else if(channels == 4){

    float4* input4c = (float4*)&input[0];
    float4* output4c = (float4*)&output[0];

    kernelSampleAndAssign(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_HORIZONTAL,fillMode,interpolationMode,make_float4(0.0,0.0,0.0,0.0));
    
  }
}

extern "C" __global__ void blurVertical(float* output,float* input, float* kernel, unsigned int kernelSize, unsigned int* dims, unsigned int fillMode,unsigned int interpolationMode)
{
  size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int height = dims[0];
  unsigned int width = dims[1];

  if (dstx >= width || dsty >= height)
      return;

  uint2 dstPoint = make_uint2(dstx,dsty);

  unsigned int channels = dims[2];


  if(channels == 1){
    kernelSampleAndAssign(input,output,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,0.0f);
  }
  else if(channels == 3){
    
    float3* input3c = (float3*)&input[0];
    float3* output3c = (float3*)&output[0];

    kernelSampleAndAssign(input3c,output3c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,make_float3(0.0,0.0,0.0));
  }
  else if(channels == 4){

    float4* input4c = (float4*)&input[0];
    float4* output4c = (float4*)&output[0];

    kernelSampleAndAssign(input4c,output4c,dims,dstPoint,kernel,kernelSize,KERNEL_1D_VERTICAL,fillMode,interpolationMode,make_float4(0.0,0.0,0.0,0.0));
    
  }
}