#include "hip/hip_runtime.h"
#include "utils.cu"

extern "C" __global__ 
void scale(float* image,float* out, float* scale, float* pivot, unsigned int* dims, unsigned int fillMode, unsigned int interpolationMode)
{
 size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
 size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

 unsigned int height = dims[0];
 unsigned int width = dims[1];
 unsigned int channels = dims[2];

 float pivotx = pivot[0] * (width-1);
 float pivoty = pivot[1] * (height-1);

 float srcx = ((float(dstx) - pivotx)/ scale[0]) + pivotx;
 float srcy = ((float(dsty) - pivoty)/ scale[1]) + pivoty;


 if (dstx >= width || dsty >= height)
    return;

  size_t outIdx = dsty*width + dstx;

  sampleAndAssign(image,out,make_float2(srcx,srcy),outIdx,dims,fillMode,interpolationMode);
  
}