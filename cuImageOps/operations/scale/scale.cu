#include "hip/hip_runtime.h"
#include "utils.cu"

extern "C" __global__ 
void scale(float* image,float* out, float* scale, float* pivot, unsigned int* dims, unsigned int fillMode, unsigned int interpolationMode)
{
 size_t dstx = blockIdx.x * blockDim.x + threadIdx.x;
 size_t dsty = blockIdx.y * blockDim.y + threadIdx.y;

 unsigned int height = dims[0];
 unsigned int width = dims[1];
 unsigned int channels = dims[2];

 float pivotx = pivot[0] * (width-1);
 float pivoty = pivot[1] * (height-1);

 float srcx = ((float(dstx) - pivotx)/ scale[0]) + pivotx;
 float srcy = ((float(dsty) - pivoty)/ scale[1]) + pivoty;


 if (dstx >= width || dsty >= height)
    return;

  size_t outIdx = dsty*width + dstx;

  if(channels == 4){
    float4* image4c = (float4*)&image[0];
    float4* out4c = (float4*)&out[0];
    out4c[outIdx] = sample2d<float4>(image4c,srcx,srcy,dims,fillMode,interpolationMode,make_float4(0.0f,0.0f,0.0f,0.0f));
  }
  else if(channels == 3){
    float3* image3c = (float3*)&image[0];
    float3* out3c = (float3*)&out[0];
    out3c[outIdx] = sample2d<float3>(image3c,srcx,srcy,dims,fillMode,interpolationMode,make_float3(0.0f,0.0f,0.0f));
  }
  else{
    out[outIdx] = sample2d<float>(image,srcx,srcy,dims,fillMode,interpolationMode,0.0f);
  }

  
}